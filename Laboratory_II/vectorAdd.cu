
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define dtype float

__global__ void VecAdd(dtype *A, dtype *B, dtype *C, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < N) {
		C[i] = A[i] + B[i];
	}
}

void printVector(dtype *v, int N) {
	for (int i = 0; i < N; i++)
		printf("%.2f\t", v[i]);
	printf("\n");
}

int main(int argc, char *argv[] ) {

	if (argc != 2) {
		printf("Usage: <N> (vector size)\n");
		return;
	}

	int N = atoi(argv[1]);
	size_t size = N * sizeof(dtype);

	dtype *h_A = (dtype *)malloc(size);
	dtype *h_B = (dtype *)malloc(size);
	dtype *h_C = (dtype *)malloc(size);

	// initialize host memory
	for (int i = 0; i < N; i++) {
		h_A[i] = rand() / (dtype)RAND_MAX;
		h_B[i] = rand() / (dtype)RAND_MAX;
	}

	printVector(h_A, N);
	printVector(h_B, N);

	dtype *d_A, *d_B, *d_C;
	hipMalloc(&d_A, size);
	hipMalloc(&d_B, size);
	hipMalloc(&d_C, size);

	// copy host memory to device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printVector(h_C, N);

	// free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}